#include "hip/hip_runtime.h"
#include "precomp.h"


// float3 operations
__device__ float3 operator+(const float3 &a, const float3 &b)
{
	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}
__device__ float3 operator-(const float3 &a, const float3 &b)
{
	return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}
__device__ float3 operator*(const float3 &a, const float &b)
{
	return make_float3(a.x * b, a.y * b, a.z * b);
}
__device__ float3 operator*(const float3 &a, const float3 &b)
{
	return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}
__device__ float3 normalize(float3 in)
{
	float mag = 1 / sqrtf(in.x * in.x + in.y * in.y + in.z*in.z);
	return make_float3(in.x * mag, in.y * mag, in.z * mag);
}
__device__ float3 cross(float3 a, float3 b)
{
	return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);
}
__device__ float dot(float3 a, float3 b)
{
	return (a.x * b.x + a.y * b.y + a.z * b.z);
}


// SSAA random number generation
__device__ uint g_seed = 0x12345678;
__device__ inline uint g_RandomUInt() { g_seed ^= g_seed << 13; g_seed ^= g_seed >> 17; g_seed ^= g_seed << 5; return g_seed; }
__device__ inline float g_RandomFloat() { return g_RandomUInt() * 2.3283064365387e-10f; }

__device__ float g_random1 = 0.529669f;
__device__ float g_random2 = 0.083422f;
__device__ float g_random3 = 0.281753f;
__device__ float g_random4 = 0.506648f;
__device__ float g_random5 = 0.438385f;
__device__ float g_random6 = 0.162733f;
__device__ float g_random7 = 0.538243f;
__device__ float g_random8 = 0.769904f;


__global__ void testkernel(float* a)
{
	int i = threadIdx.x;
	if (i > 99) return;
	a[i] = 2 * a[i];
	printf("Gpu thread %i says hi! \n", i);
}

// Adds the given color to the intermediate screen buffer
__device__ void g_addToIntermediate(g_Color* buffer, float x, float y, g_Color color)
{
	buffer[(int)x + ((int)y * SCRWIDTH)] += color;
};

// Adds a given ray to the given ray queue and updates the queue size
__device__ void addRayToQueue(float* ray, float* queue)
{
	int id = atomicInc(((uint*)queue) + 1, 0xffffffff) + 1;
	int queuesize = ((uint*)queue)[0];

	if (id > queuesize / R_SIZE)
	{
		printf("ERROR: Queue overflow. Rays exceeded the %i indices of ray queue space.\n", (int)(queuesize / R_SIZE));
	}

	int baseIndex = id * R_SIZE;

	queue[baseIndex + R_OX] = ray[0];
	queue[baseIndex + R_OY] = ray[1];
	queue[baseIndex + R_OZ] = ray[2];
	queue[baseIndex + R_DX] = ray[3];
	queue[baseIndex + R_DY] = ray[4];
	queue[baseIndex + R_DZ] = ray[5];
	queue[baseIndex + R_INOBJ] = ray[6];
	queue[baseIndex + R_REFRIND] = ray[7];
	queue[baseIndex + R_BVHTRA] = ray[8];
	queue[baseIndex + R_DEPTH] = ray[9];
	queue[baseIndex + R_PIXX] = ray[10];
	queue[baseIndex + R_PIXY] = ray[11];
	queue[baseIndex + R_ENERGY] = ray[12];
}

// Generates and returns a primary ray given the virtual screen coordinates
__device__ float* generateRayTroughVirtualScreen(float pixelx, float pixely, bool DoF, float3 position, float3 virtualScreenCornerTL, float3 virtualScreenCornerTR, float3 virtualScreenCornerBL)
{
	float3 pixelPosScaled;
	pixelPosScaled.x = pixelx / SCRWIDTH; //Scale the pixel position to be in the range 0..1
	pixelPosScaled.y = pixely / SCRHEIGHT;

	float3 DofRandomness = { 0, 0, 0 };
	if (DoF) DofRandomness = make_float3((g_RandomFloat() * 0.1 - 0.05), (g_RandomFloat() * 0.1 - 0.05), 0); //TODO: make random and maybe 7-gon instead of square?

	float3 origin = position + DofRandomness;
	//printf("ray origin: %f, %f, %f", origin.x, origin.y, origin.z);

	float3 positionOnVirtualScreen = virtualScreenCornerTL + (virtualScreenCornerTR - virtualScreenCornerTL) * pixelPosScaled.x + (virtualScreenCornerBL - virtualScreenCornerTL) * pixelPosScaled.y;
	float3 direction = normalize(positionOnVirtualScreen - origin);

	float* ray = new float[R_SIZE];
	ray[0] = origin.x;
	ray[1] = origin.y;
	ray[2] = origin.z;
	ray[3] = direction.x;
	ray[4] = direction.y;
	ray[5] = direction.z;
	//float ray[6] = { origin.x, origin.y, origin.z, direction.x, direction.y, direction.z };

	return ray;
}

// Generates and collects primary rays in the given ray queue
__global__ void GeneratePrimaryRay(float* rayQueue, bool DoF, float3 position, float3 virtualScreenCornerTL, float3 virtualScreenCornerTR, float3 virtualScreenCornerBL, bool SSAA)
{
	uint numRays = SCRWIDTH * SCRHEIGHT;
	uint raynum = atomicInc(((uint*)rayQueue) + 2, 0xffffffff);

	while (raynum < numRays) {
		int pixelx = raynum % SCRWIDTH;
		int pixely = raynum / SCRWIDTH;
		//int pixelx = threadIdx.x;
		//int pixely = blockIdx.x;

		//printf("id: %i, x: %i, y: %i \n", raynum, pixelx, pixely);

		if (pixelx > SCRWIDTH || pixely > SCRHEIGHT) printf("wtf");

		//Generate the ray
		if (SSAA) {
			float* ray1 = generateRayTroughVirtualScreen((float)pixelx + g_random1, (float)pixely + g_random2, DoF, position, virtualScreenCornerTL, virtualScreenCornerTR, virtualScreenCornerBL);

			ray1[R_INOBJ] = 0;
			ray1[R_REFRIND] = 1.0f;
			ray1[R_BVHTRA] = 0;
			ray1[R_DEPTH] = 0;
			ray1[R_PIXX] = pixelx;
			ray1[R_PIXY] = pixely;
			ray1[R_ENERGY] = 0.25f;

			addRayToQueue(ray1, rayQueue);
			delete ray1;

			float* ray2 = generateRayTroughVirtualScreen((float)pixelx + g_random3, (float)pixely + g_random4, DoF, position, virtualScreenCornerTL, virtualScreenCornerTR, virtualScreenCornerBL);

			ray2[R_INOBJ] = 0;
			ray2[R_REFRIND] = 1.0f;
			ray2[R_BVHTRA] = 0;
			ray2[R_DEPTH] = 0;
			ray2[R_PIXX] = pixelx;
			ray2[R_PIXY] = pixely;
			ray2[R_ENERGY] = 0.25f;

			addRayToQueue(ray2, rayQueue);
			delete ray2;

			float* ray3 = generateRayTroughVirtualScreen((float)pixelx + g_random5, (float)pixely + g_random6, DoF, position, virtualScreenCornerTL, virtualScreenCornerTR, virtualScreenCornerBL);

			ray3[R_INOBJ] = 0;
			ray3[R_REFRIND] = 1.0f;
			ray3[R_BVHTRA] = 0;
			ray3[R_DEPTH] = 0;
			ray3[R_PIXX] = pixelx;
			ray3[R_PIXY] = pixely;
			ray3[R_ENERGY] = 0.25f;

			addRayToQueue(ray3, rayQueue);
			delete ray3;

			float* ray4 = generateRayTroughVirtualScreen((float)pixelx + g_random7, (float)pixely + g_random8, DoF, position, virtualScreenCornerTL, virtualScreenCornerTR, virtualScreenCornerBL);

			ray4[R_INOBJ] = 0;
			ray4[R_REFRIND] = 1.0f;
			ray4[R_BVHTRA] = 0;
			ray4[R_DEPTH] = 0;
			ray4[R_PIXX] = pixelx;
			ray4[R_PIXY] = pixely;
			ray4[R_ENERGY] = 0.25f;

			addRayToQueue(ray4, rayQueue);
			delete ray4;

		}
		else {
			float* ray = generateRayTroughVirtualScreen(pixelx, pixely, DoF, position, virtualScreenCornerTL, virtualScreenCornerTR, virtualScreenCornerBL);

			ray[R_INOBJ] = 0;
			ray[R_REFRIND] = 1.0f;
			ray[R_BVHTRA] = 0;
			ray[R_DEPTH] = 0;
			ray[R_PIXX] = pixelx;
			ray[R_PIXY] = pixely;
			ray[R_ENERGY] = 1.0f;

			addRayToQueue(ray, rayQueue);
			delete ray;
		}
		raynum = atomicInc(((uint*)rayQueue) + 2, 0xffffffff);
	}

}

// Finds a collision with the triangle. Returns a collision with t = -1 if none were found.
__device__ g_Collision g_intersectTriangle(int i, float* ray_ptr, float * triangles, bool isShadowRay = false)
{
	int baseindex = i * FLOATS_PER_TRIANGLE;

	float3 v0 = {
		triangles[baseindex + T_V0X],
		triangles[baseindex + T_V0Y],
		triangles[baseindex + T_V0Z] };
	float3 v1 = {
		triangles[baseindex + T_V1X],
		triangles[baseindex + T_V1Y],
		triangles[baseindex + T_V1Z] };
	float3 v2 = {
		triangles[baseindex + T_V2X],
		triangles[baseindex + T_V2Y],
		triangles[baseindex + T_V2Z] };
	float3 e0 = {
		triangles[baseindex + T_E0X],
		triangles[baseindex + T_E0Y],
		triangles[baseindex + T_E0Z] };
	float3 e1 = {
		triangles[baseindex + T_E1X],
		triangles[baseindex + T_E1Y],
		triangles[baseindex + T_E1Z] };
	float3 e2 = {
		triangles[baseindex + T_E2X],
		triangles[baseindex + T_E2Y],
		triangles[baseindex + T_E2Z] };
	float3 N = { triangles[baseindex + T_NX],
		triangles[baseindex + T_NY],
		triangles[baseindex + T_NZ] };

	float3 direction = { ray_ptr[R_DX], ray_ptr[R_DY], ray_ptr[R_DZ] };
	float3 origin = { ray_ptr[R_OX], ray_ptr[R_OY], ray_ptr[R_OZ] };

	float D = triangles[baseindex + T_D];
	

	g_Collision collision;
	collision.t = -1;

	float NdotR = dot(direction, N);
	if (NdotR == 0) return collision; //Ray parrallel to plane, would cause division by 0

	float t = -(dot(origin, N) + D) / (NdotR);

	//From https://www.scratchapixel.com/lessons/3d-basic-rendering/ray-tracing-rendering-a-triangle/ray-triangle-intersection-geometric-solution
	if (t > 0)
	{
		float3 P = origin + direction * t;
		if (dot(N, cross(e0, (P - v0))) > 0 && dot(N, cross(e1, (P - v1))) > 0 && dot(N, cross(e2, (P - v2))) > 0)
		{
			//Collision
			collision.t = t;

			if (isShadowRay) {
				return collision;
			}

			collision.R = triangles[baseindex + T_COLORR];
			collision.G = triangles[baseindex + T_COLORG];
			collision.B = triangles[baseindex + T_COLORB];
			//collision.other = triangles + baseindex;
			collision.refraction = triangles[baseindex + T_REFRACTION];
			collision.specularity = triangles[baseindex + T_SPECULARITY];
			if (NdotR > 0) collision.N = N * -1;
			else collision.N = N;
			collision.Pos = P;
			return collision;
		}
	}
	return collision;
}

// Checks if the ray intersects the BVH node. Returns 'tmin' if it does, and returns -99999 otherwise.
__device__ float g_IntersectAABB(float* ray_ptr, float* BVHNode)
{
	float xmin = BVHNode[B_AABB_MINX];
	float xmax = BVHNode[B_AABB_MAXX];
	float ymin = BVHNode[B_AABB_MINY];
	float ymax = BVHNode[B_AABB_MAXY];
	float zmin = BVHNode[B_AABB_MINZ];
	float zmax = BVHNode[B_AABB_MAXZ];

	float dirX = ray_ptr[R_DX];
	float dirY = ray_ptr[R_DY];
	float dirZ = ray_ptr[R_DZ];
	float OX = ray_ptr[R_OX];
	float OY = ray_ptr[R_OY];
	float OZ = ray_ptr[R_OZ];

	float invDirX = 1 / dirX;
	float tmin = (xmin - OX) * invDirX;
	float tmax = (xmax - OX) * invDirX;

	if (tmin > tmax) { 
		float temp = tmin;
		tmin = tmax;
		tmax = temp;
	}

	float invDirY = 1 / dirY;
	float tymin = (ymin - OY) * invDirY;
	float tymax = (ymax - OY) * invDirY;

	if (tymin > tymax) {
		float temp = tymin;
		tymin = tymax;
		tymax = temp;

		//swap(tymin, tymax);
	}

	if ((tmin > tymax) || (tymin > tmax))
		return -99999;

	tmin = max(tmin, tymin);
	tmax = min(tymax, tmax);

	float invDirZ = 1 / dirZ;
	float tzmin = (zmin - OZ) * invDirZ;
	float tzmax = (zmax - OZ) * invDirZ;

	if (tzmin > tzmax) {
		float temp = tzmin;
		tzmin = tzmax;
		tzmax = temp;

		//swap(tzmin, tzmax);
	}

	if ((tmin > tzmax) || (tzmin > tmax))
		return -99999;

	tmin = max(tmin, tzmin);
	tmax = min(tzmax, tmax);

	if (tmax < 0) return -99999;

	return tmin;
}

// Recursively traverses the BVH tree from the given node to find a collision. Returns a collision with t = -1 if none were found.
__device__ g_Collision g_TraverseBVHNode(float* ray_ptr, float* pool, uint* orderedIndices, float* scene, int index, int* stack, float* stackAABBEntrypoints)
{
	g_Collision closest;
	closest.t = -1;

	ray_ptr[R_BVHTRA]++;
	int count = pool[index + B_COUNT];
		// If leaf
	if (count != 0)
	{
		float closestdist = 0xffffff;


		// Find closest collision
		for (int i = 0; i < pool[index + B_COUNT]; i++)
		{
			//Collision collision = scene[orderedIndices[node->leftFirst + i]]->Intersect(*ray);
			int triangleindex = orderedIndices[(int)pool[index + B_LEFTFIRST] + i];

			g_Collision collision = g_intersectTriangle(triangleindex, ray_ptr, scene);
			float dist = collision.t;
			if (dist != -1 && dist < closestdist)
			{
				//Collision. Check if closest
				closest = collision;
				closestdist = dist;
			}
		}
		//printf("leaf: collision at %f \n", closest.t);
		return closest;
	}
	// If node
	else
	{
		//This is not a leaf
		// Check both children and return the closest collision if both intersected
		int leftchild = pool[(int)index + B_LEFTFIRST];
		int rightchild = leftchild + B_SIZE;

		float tleft = g_IntersectAABB(ray_ptr, pool + leftchild);
		float tright = g_IntersectAABB(ray_ptr, pool + rightchild);

		int flip = 0;

		int baseIndexNear = leftchild;
		int baseIndexFar = rightchild;

		float tEntryFarNode = tright;
		float tEntryNearNode = tleft;
		if (tright < tleft && tright > -99999) {
			baseIndexNear = rightchild;
			baseIndexFar = leftchild;
			tEntryFarNode = tleft;
			tEntryNearNode = tright;
		};

		if (tEntryNearNode > -99999) {
			int stackindex = ++stack[0];
			if (stackindex >= 32) printf("stack too small!. index: %i \n", stackindex);
			else {
				stack[stackindex] = baseIndexNear;
				stackAABBEntrypoints[stackindex] = tEntryNearNode;
				//printf("Added %i to stack location %i. This is the near child of %i \n", baseIndexNear, stackindex, index);

			}
		}
		if (tEntryFarNode > -99999) {
			int stackindex = ++stack[0];
			if (stackindex >= 32) printf("stack too small!. index: %i \n", stackindex);

			else {
				stack[stackindex] = baseIndexFar;
				stackAABBEntrypoints[stackindex] = tEntryFarNode;
				//printf("Added %i to stack location %i. Right child of %i \n", baseIndexFar, stackindex, index);
			}
		}

		return closest;
	}

	return closest;
}

// Finds the first geometry collision in its path. Returns a collision with t = -1 if none were found.
__device__ g_Collision g_nearestCollision(float* ray_ptr, bool use_bvh, int numGeometries, float* triangles, float* BVH, uint* orderedIndices)
{
	if (use_bvh)
	{
		int* stack = new int[32];
		float* aabbEntryPoints = new float[32];
		aabbEntryPoints[2] = -5000.0f;

		stack[0] = 1; //count, next one to evaluate;
		stack[1] = 0; //Root node

		g_Collision closest;
		closest.t = -1;

		while (stack[0] > 0)
		{
			int next = stack[0]--;
			//printf("next: stack[%i]: %i. AABB entrypoint: %f \n", next, stack[next], aabbEntryPoints[next]);

			if (closest.t != -1 && closest.t < aabbEntryPoints[next]) {
				continue;
			}

			g_Collision newcollision = g_TraverseBVHNode(ray_ptr, BVH, orderedIndices, triangles, stack[next], stack, aabbEntryPoints);

			if ((newcollision.t != -1 && newcollision.t < closest.t) || closest.t == -1) {
				closest = newcollision;
				//printf("closest t now %f \n", closest.t);
			}
		}
		delete stack;
		delete aabbEntryPoints;
		return closest;
	}
	else
	{
		float closestdist = 0xffffff;
		g_Collision closest;
		closest.t = -1;

		//Loop over all primitives to find the closest collision
		for (int i = 0; i < numGeometries; i++)
		{
			g_Collision collision = g_intersectTriangle(i, ray_ptr, triangles);
			float dist = collision.t;
			if (dist != -1 && dist < closestdist)
			{
				//Collision. Check if closest
				closest = collision;
				closestdist = dist;
			}
		}
		return closest;
	}
}

// Generates and collects the nearest geometry intersections for the given ray queue
__global__ void g_findCollisions(float* triangles, int numtriangles, float* rayQueue, void* collisions, bool useBVH, float* BVH, unsigned int* orderedIndices)
{
	uint numRays = ((uint*)rayQueue)[1];
	uint id = atomicInc(((uint*)rayQueue) + 3, 0xffffffff) + 1;

	while (id <= numRays)
	{
		if (id != 0)
		{
			float* rayptr = rayQueue + (id * R_SIZE);
			g_Collision collision = g_nearestCollision(rayptr, useBVH, numtriangles, triangles, BVH, orderedIndices);
			((g_Collision*)collisions)[id] = collision;
		}
		id = atomicInc(((uint*)rayQueue) + 3, 0xffffffff) + 1;
	}
}

// Checks for geometry intersections with the given shadow ray queue, and adds their energy to the intermediate screen buffer if unoccluded
__global__ void g_TraceShadowRay(float* shadowrays, int rayIndex, bool use_bvh, float* BVH, unsigned int* orderedIndices, int numGeometries, float* scene, g_Color* intermediate, int bvhStackCapacity)
{
	int baseIndex = rayIndex * SR_SIZE;
	float maxt = shadowrays[baseIndex + SR_MAXT];
	bool collided = false;

	// Extracting shadow ray from ray queue
	float shadowray[R_SIZE] = {
		shadowrays[baseIndex + SR_OX],
		shadowrays[baseIndex + SR_OY],
		shadowrays[baseIndex + SR_OZ],
		shadowrays[baseIndex + SR_DX],
		shadowrays[baseIndex + SR_DY],
		shadowrays[baseIndex + SR_DZ]
	};

	if (use_bvh)
	{
		// Initializing stack
		float* AABBEntryPoints = new float[bvhStackCapacity];
		int* stack = new int[bvhStackCapacity];
		stack[0] = 1; //count, next one to evaluate;
		stack[1] = 0; //root node

		// Traversing BVH
		while (stack[0] > 0)
		{
			int next = stack[0]--;
			g_Collision newcollision = g_TraverseBVHNode(shadowray, BVH, orderedIndices, scene, stack[next], stack, AABBEntryPoints);
			if (newcollision.t > 0 && newcollision.t < maxt) break; // collision: light source is occluded
		}

		// Cleaning up
		delete stack;
		delete AABBEntryPoints;
	}
	else
	{
		// NOTE: Not using a BVH on the GPU will cause an automatic kernel shutdown after 5 seconds when there are too many triangles
		for (int i = 0; i < numGeometries; i++)
		{
			g_Collision shadowcollision = g_intersectTriangle(i, shadowray, scene, true);
			if (shadowcollision.t != -1 && shadowcollision.t < maxt) return; // collision: light source is occluded
		}
	}

	// Adding the unoccluded ray to the intermediate screen buffer
	g_Color toadd = g_Color(shadowrays[baseIndex + SR_R], shadowrays[baseIndex + SR_G], shadowrays[baseIndex + SR_B]);
	g_addToIntermediate(intermediate, shadowrays[baseIndex + SR_PIXX], shadowrays[baseIndex + SR_PIXY], toadd);
}

__device__ float3 g_reflect(float3 D, float3 N)
{
	return D - N * (2 * (dot(D, N)));
}

__device__ float sqrLentgh(float3 a) //Not my typo. It was in the template and I'm keeping it to keep it consistent
{
	return a.x * a.x + a.y * a.y + a.z * a.z;
}

__device__ void g_addShadowRayToQueue(float3 ori, float3 dir, float R, float G, float B, float maxt, float pixelX, float pixelY, float* queue)
{
	int id = atomicInc(((uint*)queue) + 1, 0xffffffff) + 1;
	int queuesize = ((uint*)queue)[0];

	// array is full
	if (id > queuesize / SR_SIZE)
	{
		printf("ERROR: Queue overflow. Rays exceeded the %d indices of shadowray queue space.\n", queuesize / R_SIZE);
	}

	// adding ray to array
	int index = id * SR_SIZE; //Keep the first entry in the queue free, to save some metadata there (queuesize, currentCount)
	queue[index + SR_OX] = (float)ori.x;
	queue[index + SR_OY] = (float)ori.y;
	queue[index + SR_OZ] = (float)ori.z;
	queue[index + SR_DX] = (float)dir.x;
	queue[index + SR_DY] = (float)dir.y;
	queue[index + SR_DZ] = (float)dir.z;
	queue[index + SR_R] = R;
	queue[index + SR_G] = G;
	queue[index + SR_B] = B;

	queue[index + SR_MAXT] = maxt;
	queue[index + SR_PIXX] = pixelX;
	queue[index + SR_PIXY] = pixelY;
}

__device__ void g_TraceRay(float* rays, int ray, g_Collision* collisions, float* newRays, float* shadowRays, bool bvhdebug, g_Color* intermediate, int numLights, float* lightPos, g_Color* lightColor)
{
	//printf("traceray");

	float* ray_ptr = rays + (ray * R_SIZE);
	// unpacking ray pointer
	float3 direction = make_float3(ray_ptr[R_DX], ray_ptr[R_DY], ray_ptr[R_DZ]);
	bool inobj = ray_ptr[R_INOBJ];
	float refind = ray_ptr[R_REFRIND];
	float rdepth = ray_ptr[R_DEPTH];
	float pixelx = ray_ptr[R_PIXX];
	float pixely = ray_ptr[R_PIXY];
	float energy = ray_ptr[R_ENERGY];

	// Basecase
	if (ray_ptr[R_DEPTH] > MAX_RECURSION_DEPTH)
	{
		//return 0x000000;
		return;
	}

	// Collision detection
	g_Collision collision = collisions[ray];
	if (bvhdebug) {
		g_addToIntermediate(intermediate, pixelx, pixely, (g_Color(255, 0, 0) * ray_ptr[R_BVHTRA]) << 3);;
		return;
	}

	// if ray collides
	if (collision.t > 0)
	{
		// if opaque
		if (collision.refraction == 0.0f)
		{
			g_Color albedo, reflection;
			float specularity = collision.specularity;

			// diffuse aspect
			if (specularity < 1.0f)
			{
				//Generate shadow rays
				for (int light = 0; light < numLights; light++)
				{
					float3 lightPosition = make_float3(lightPos[light * 3 + 0], lightPos[light * 3 + 1], lightPos[light * 3 + 2]);
					float3 direction = normalize(lightPosition - collision.Pos);
					float3 origin = collision.Pos + ( direction * 0.00025f); //move away a little bit from the surface, to avoid self-collision in the outward direction.
					float maxt = (lightPos[light * 3 + 0] - collision.Pos.x) / direction.x; //calculate t where the shadowray hits the light source. Because we don't want to count collisions that are behind the light source.


					float3 collisioncolor = make_float3(collision.R, collision.G, collision.B);
					float3 lightColorAsFloat3 = make_float3(lightColor[light].R, lightColor[light].G, lightColor[light].B);

					float3 shadowRayEnergy = collisioncolor * energy * (1 - specularity) * lightColorAsFloat3 * (max(0.0f, dot(collision.N, direction)) * INV4PI / sqrLentgh(lightPosition - collision.Pos));

					g_addShadowRayToQueue(origin, direction, shadowRayEnergy.x, shadowRayEnergy.y, shadowRayEnergy.z, maxt, pixelx, pixely, shadowRays);
				}
			}

			// specular aspect
			if (specularity > 0)
			{
				float3 newdirection = g_reflect(direction, collision.N);
				float3 newOrigin = collision.Pos + newdirection * 0.00001f;
				float* newray = new float[R_SIZE];
				newray[R_OX] = newOrigin.x;
				newray[R_OY] = newOrigin.y;
				newray[R_OZ] = newOrigin.z;
				newray[R_DX] = newdirection.x;
				newray[R_DY] = newdirection.y;
				newray[R_DZ] = newdirection.z;
				newray[R_INOBJ] = (float)inobj;
				newray[R_REFRIND] = refind;
				newray[R_BVHTRA] = 0;
				newray[R_DEPTH] = rdepth + 1;
				newray[R_PIXX] = pixelx;
				newray[R_PIXY] = pixely;
				newray[R_ENERGY] = energy * specularity;

				addRayToQueue(newray, newRays);
				delete newray;
				
			}
		}
		// if transparent
		else
		{
			float n1, n2;
			if (inobj) n1 = refind, n2 = 1.0f;
			else				n1 = refind, n2 = collision.refraction;
			float transition = n1 / n2;
			float costheta = dot(collision.N, direction * -1);
			float k = 1 - (transition * transition) * (1.0f - (costheta * costheta));

			float Fr;
			if (k < 0)
			{
				// total internal reflection
				Fr = 1;
			}
			else
			{
				float ndiff = n1 - n2;
				float nsumm = n1 + n2;
				float temp = ndiff / nsumm;
				float R0 = temp * temp;
				Fr = R0 + (1.0f - R0) * powf(1.0f - costheta, 5.0f);
			}

			// Fresnel reflection (Schlick's approximation)
			g_Color reflection, refraction;
			if (Fr > 0.0f)
			{
				float3 newdirection = g_reflect(direction, collision.N);


				float3 newOrigin = collision.Pos + newdirection * 0.00001f;
				float* newray = new float[R_SIZE];
				newray[R_OX] = newOrigin.x;
				newray[R_OY] = newOrigin.y;
				newray[R_OZ] = newOrigin.z;
				newray[R_DX] = newdirection.x;
				newray[R_DY] = newdirection.y;
				newray[R_DZ] = newdirection.z;
				newray[R_INOBJ] = (float)inobj;
				newray[R_REFRIND] = refind;
				newray[R_BVHTRA] = 0;
				newray[R_DEPTH] = rdepth + 1;
				newray[R_PIXX] = pixelx;
				newray[R_PIXY] = pixely;
				newray[R_ENERGY] = energy * Fr;

				addRayToQueue(newray, newRays);
				delete newray;

			}

			// Snell refraction
			if (Fr < 1.0f)
			{
				float3 newdirection = direction * transition + collision.N * (transition * costheta - sqrt(k));
				float3 newOrigin = collision.Pos + newdirection * 0.00001f;
				float* newray = new float[R_SIZE];
				newray[R_OX] = newOrigin.x;
				newray[R_OY] = newOrigin.y;
				newray[R_OZ] = newOrigin.z;
				newray[R_DX] = newdirection.x;
				newray[R_DY] = newdirection.y;
				newray[R_DZ] = newdirection.z;
				newray[R_INOBJ] = (float)inobj;
				newray[R_REFRIND] = refind;
				newray[R_BVHTRA] = 0;
				newray[R_DEPTH] = rdepth + 1;
				newray[R_PIXX] = pixelx;
				newray[R_PIXY] = pixely;
				newray[R_ENERGY] = energy * (1 - Fr);

				addRayToQueue(newray, newRays);
				delete newray;


				/* // TODO: Beer's law (and mirror albedo) requires ray.energy to be a Color rather than a float
				// Beer's law
				if (ray.mediumRefractionIndex != 1.0f && collision.colorAt.to_uint() != 0xffffff)
				{
					float distance = collision.t;

					vec3 a = vec3((float)(256 - collision.colorAt.R) / 256.0f, (float)(256 - collision.colorAt.G) / 256.0f, (float)(256 - collision.colorAt.B) / 256.0f);

					refraction.R *= exp(-a.x * distance);
					refraction.G *= exp(-a.y * distance);
					refraction.B *= exp(-a.z * distance);
				}
				*/
			}
		}
	}
	// if no collision
	else
	{
		//TODO: implement skybox
		g_addToIntermediate(intermediate, pixelx, pixely, (g_Color(40, 20, 150) << 8) * energy);
	}
}

__global__ void g_Tracerays(float* rayQueue, void* collisions, float* newRays, float* shadowRays, bool bvhdebug, g_Color* intermediate, int numLights, float* lightPos, g_Color* lightColor)
{
	uint numRays = ((uint*)rayQueue)[1];
	uint id = atomicInc(((uint*)rayQueue) + 4, 0xffffffff) + 1;

	while (id <= numRays)
	{
		if (id != 0)
		{
			float* rayptr = rayQueue + (id * R_SIZE);
			g_TraceRay(rayQueue, id, (g_Collision*)collisions, newRays, shadowRays, bvhdebug, intermediate, numLights, lightPos, lightColor);
		}
		id = atomicInc(((uint*)rayQueue) + 4, 0xffffffff) + 1;
	}
}
